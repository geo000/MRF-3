#include "hip/hip_runtime.h"
#include"cuda_utility.cuh"

 hipEvent_t start_timer;
 hipEvent_t end_timer;

#define CREATE_CUDA_TIMER						\
CUDA_CHECK(hipEventCreate(&start_timer));		\
CUDA_CHECK(hipEventCreate(&end_timer));		\
CUDA_CHECK(hipEventRecord(start_timer,0));		\

#define END_CUDA_TIMER							\
CUDA_CHECK(hipEventRecord(end_timer,0));		\
CUDA_CHECK(hipEventSynchronize(stop));			\
float elapsedTime=0;							\
CUDA_CHECK(hipEventElapsedTime(&elapsedTime,start_timer,end_timer));\
printf("Total time : %3.1f ms\n",elapsedTime);	\
CUDA_CHECK(hipEventDestroy(start_timer));		\
CUDA_CHECK(hipEventDestroy(end_timer));		\

namespace CUDA{

	__global__ void kernel_get_points(const uchar** pointMask, points_pointer** points,const int pixel_num)
	{
		int offset = threadIdx.x + threadIdx.y *blockDim.x;

		if (offset >= pixel_num)  return;

		uchar ch = pointMask[blockIdx.x][offset];
		if (ch < MAX_VALUE){



		}
		
	}
	// 
	__global__ void get_scribble_points(const MatArray& pointsMask, const MatArray& slicPointsMask, PointsArrays& points, PointsArrays& slicPoints){



	}



}