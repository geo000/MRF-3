#include "hip/hip_runtime.h"
#include"cuda_utility.cuh"

 hipEvent_t start_timer;
 hipEvent_t end_timer;

//extern texture<ushort4, hipTextureType3D, hipReadModeElementType> device_textures3D;

#define CREATE_CUDA_TIMER						\
CUDA_CHECK(hipEventCreate(&start_timer));		\
CUDA_CHECK(hipEventCreate(&end_timer));		\
CUDA_CHECK(hipEventRecord(start_timer,0));		\

#define END_CUDA_TIMER							\
CUDA_CHECK(hipEventRecord(end_timer,0));		\
CUDA_CHECK(hipEventSynchronize(stop));			\
float elapsedTime=0;							\
CUDA_CHECK(hipEventElapsedTime(&elapsedTime,start_timer,end_timer));\
printf("Total time : %3.1f ms\n",elapsedTime);	\
CUDA_CHECK(hipEventDestroy(start_timer));		\
CUDA_CHECK(hipEventDestroy(end_timer));		\

namespace CUDA{

	
	__global__ void kernel_get_points(const uchar* pointMask, thrust::device_vector<device_point>& device_points_vec, const int pixel_num)
	{
		int offset = threadIdx.x + threadIdx.y *blockDim.x;

		if (offset >= pixel_num)  return;

		uchar ch = pointMask[offset];
		if (ch < MAX_VALUE){
			 


		}
		
	}
	// 
	void get_scribble_points(const MatArray& pointsMask, const MatArray& slicPointsMask, PointsArrays& points, PointsArrays& slicPoints){


		//get basic infos
		int label_num = pointsMask.size();
		
		CHECK_GT(label_num, 0) << "Label_num must greater to 0.\n";

		int width  = pointsMask[0].cols;	// width	->     cols   -   x
		int height = pointsMask[0].rows;	// height	->     rows   -   y





	}



}