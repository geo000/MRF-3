#include "hip/hip_runtime.h"
#include"cuda_sift.cuh"
//**************************************constant value

__constant__ float d_Threshold[2];
__constant__ float d_Scales[8], d_Factor;
__constant__ float d_EdgeLimit;
__constant__ int d_MaxNumPoints;

__device__ unsigned int d_PointCounter[1];
__constant__ float d_Kernel1[5];
__constant__ float d_Kernel2[12 * 16];


//**************************************

int iDivUp(int a, int b) { return (a%b != 0) ? (a / b + 1) : (a / b); }
int iDivDown(int a, int b) { return a / b; }
int iAlignUp(int a, int b) { return (a%b != 0) ? (a - a%b + b) : a; }
int iAlignDown(int a, int b) { return a - a%b; }

void CudaImage::Allocate(int w, int h, int p, bool host, float *devmem, float *hostmem)
{
	width = w;
	height = h;
	pitch = p;
	d_data = devmem;
	h_data = hostmem;
	t_data = NULL;
	if (devmem == NULL) {
		safeCall(hipMallocPitch((void **)&d_data, (size_t*)&pitch, (size_t)(sizeof(float)*width), (size_t)height));
		pitch /= sizeof(float);
		if (d_data == NULL)
			printf("Failed to allocate device data\n");
		d_internalAlloc = true;
	}
	if (host && hostmem == NULL) {
		h_data = (float *)malloc(sizeof(float)*pitch*height);
		h_internalAlloc = true;
	}
}

CudaImage::CudaImage() :
width(0), height(0), d_data(NULL), h_data(NULL), t_data(NULL), d_internalAlloc(false), h_internalAlloc(false)
{

}

CudaImage::~CudaImage()
{
	if (d_internalAlloc && d_data != NULL)
		safeCall(hipFree(d_data));
	d_data = NULL;
	if (h_internalAlloc && h_data != NULL)
		free(h_data);
	h_data = NULL;
	if (t_data != NULL)
		safeCall(hipFreeArray((hipArray *)t_data));
	t_data = NULL;
}

double CudaImage::Download()
{
	TimerGPU timer(0);
	int p = sizeof(float)*pitch;
	if (d_data != NULL && h_data != NULL)
		safeCall(hipMemcpy2D(d_data, p, h_data, sizeof(float)*width, sizeof(float)*width, height, hipMemcpyHostToDevice));
	double gpuTime = timer.read();
#ifdef VERBOSE
	printf("Download time =               %.2f ms\n", gpuTime);
#endif
	return gpuTime;
}

double CudaImage::Readback()
{
	TimerGPU timer(0);
	int p = sizeof(float)*pitch;
	safeCall(hipMemcpy2D(h_data, sizeof(float)*width, d_data, p, sizeof(float)*width, height, hipMemcpyDeviceToHost));
	double gpuTime = timer.read();
#ifdef VERBOSE
	printf("Readback time =               %.2f ms\n", gpuTime);
#endif
	return gpuTime;
}

double CudaImage::InitTexture()
{
	TimerGPU timer(0);
	hipChannelFormatDesc t_desc = hipCreateChannelDesc<float>();
	safeCall(hipMallocArray((hipArray **)&t_data, &t_desc, pitch, height));
	if (t_data == NULL)
		printf("Failed to allocated texture data\n");
	double gpuTime = timer.read();
#ifdef VERBOSE
	printf("InitTexture time =            %.2f ms\n", gpuTime);
#endif
	return gpuTime;
}

double CudaImage::CopyToTexture(CudaImage &dst, bool host)
{
	if (dst.t_data == NULL) {
		printf("Error CopyToTexture: No texture data\n");
		return 0.0;
	}
	if ((!host || h_data == NULL) && (host || d_data == NULL)) {
		printf("Error CopyToTexture: No source data\n");
		return 0.0;
	}
	TimerGPU timer(0);
	if (host)
		safeCall(hipMemcpyToArray((hipArray *)dst.t_data, 0, 0, h_data, sizeof(float)*pitch*dst.height, hipMemcpyHostToDevice));
	else
		safeCall(hipMemcpyToArray((hipArray *)dst.t_data, 0, 0, d_data, sizeof(float)*pitch*dst.height, hipMemcpyDeviceToDevice));
	safeCall(hipDeviceSynchronize());
	double gpuTime = timer.read();
#ifdef VERBOSE
	printf("CopyToTexture time =          %.2f ms\n", gpuTime);
#endif
	return gpuTime;
}

//**********************************************
///////////////////////////////////////////////////////////////////////////////
// Lowpass filter an subsample image
///////////////////////////////////////////////////////////////////////////////
__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
	__shared__ float inrow[SCALEDOWN_W + 4];
	__shared__ float brow[5 * (SCALEDOWN_W / 2)];
	__shared__ int yRead[SCALEDOWN_H + 4];
	__shared__ int yWrite[SCALEDOWN_H + 4];
#define dx2 (SCALEDOWN_W/2)
	const int tx = threadIdx.x;
	const int tx0 = tx + 0 * dx2;
	const int tx1 = tx + 1 * dx2;
	const int tx2 = tx + 2 * dx2;
	const int tx3 = tx + 3 * dx2;
	const int tx4 = tx + 4 * dx2;
	const int xStart = blockIdx.x*SCALEDOWN_W;
	const int yStart = blockIdx.y*SCALEDOWN_H;
	const int xWrite = xStart / 2 + tx;
	const float *k = d_Kernel1;
	if (tx<SCALEDOWN_H + 4) {
		int y = yStart + tx - 1;
		y = (y<0 ? 0 : y);
		y = (y >= height ? height - 1 : y);
		yRead[tx] = y*pitch;
		yWrite[tx] = (yStart + tx - 4) / 2 * newpitch;
	}
	__syncthreads();
	int xRead = xStart + tx - 2;
	xRead = (xRead<0 ? 0 : xRead);
	xRead = (xRead >= width ? width - 1 : xRead);
	for (int dy = 0; dy<SCALEDOWN_H + 4; dy += 5) {
		inrow[tx] = d_Data[yRead[dy + 0] + xRead];
		__syncthreads();
		if (tx<dx2)
			brow[tx0] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) + k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) + k[2] * inrow[2 * tx + 2];
		__syncthreads();
		if (tx<dx2 && dy >= 4 && !(dy & 1))
			d_Result[yWrite[dy + 0] + xWrite] = k[2] * brow[tx2] + k[0] * (brow[tx0] + brow[tx4]) + k[1] * (brow[tx1] + brow[tx3]);
		if (dy<(SCALEDOWN_H + 3)) {
			inrow[tx] = d_Data[yRead[dy + 1] + xRead];
			__syncthreads();
			if (tx<dx2)
				brow[tx1] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) + k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) + k[2] * inrow[2 * tx + 2];
			__syncthreads();
			if (tx<dx2 && dy >= 3 && (dy & 1))
				d_Result[yWrite[dy + 1] + xWrite] = k[2] * brow[tx3] + k[0] * (brow[tx1] + brow[tx0]) + k[1] * (brow[tx2] + brow[tx4]);
		}
		if (dy<(SCALEDOWN_H + 2)) {
			inrow[tx] = d_Data[yRead[dy + 2] + xRead];
			__syncthreads();
			if (tx<dx2)
				brow[tx2] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) + k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) + k[2] * inrow[2 * tx + 2];
			__syncthreads();
			if (tx<dx2 && dy >= 2 && !(dy & 1))
				d_Result[yWrite[dy + 2] + xWrite] = k[2] * brow[tx4] + k[0] * (brow[tx2] + brow[tx1]) + k[1] * (brow[tx3] + brow[tx0]);
		}
		if (dy<(SCALEDOWN_H + 1)) {
			inrow[tx] = d_Data[yRead[dy + 3] + xRead];
			__syncthreads();
			if (tx<dx2)
				brow[tx3] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) + k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) + k[2] * inrow[2 * tx + 2];
			__syncthreads();
			if (tx<dx2 && dy >= 1 && (dy & 1))
				d_Result[yWrite[dy + 3] + xWrite] = k[2] * brow[tx0] + k[0] * (brow[tx3] + brow[tx2]) + k[1] * (brow[tx4] + brow[tx1]);
		}
		if (dy<SCALEDOWN_H) {
			inrow[tx] = d_Data[yRead[dy + 4] + xRead];
			__syncthreads();
			if (tx<dx2)
				brow[tx4] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) + k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) + k[2] * inrow[2 * tx + 2];
			__syncthreads();
			if (tx<dx2 && !(dy & 1))
				d_Result[yWrite[dy + 4] + xWrite] = k[2] * brow[tx1] + k[0] * (brow[tx4] + brow[tx3]) + k[1] * (brow[tx0] + brow[tx2]);
		}
		__syncthreads();
	}
}


__global__ void ExtractSiftDescriptors(hipTextureObject_t texObj, SiftPoint *d_sift, int fstPts, float subsampling)
{
	__shared__ float gauss[16];
	__shared__ float buffer[128];
	__shared__ float sums[128];

	const int tx = threadIdx.x; // 0 -> 16
	const int ty = threadIdx.y; // 0 -> 16
	const int idx = ty * 16 + tx;
	const int bx = blockIdx.x + fstPts;  // 0 -> numPts
	if (ty == 0)
		gauss[tx] = exp(-(tx - 7.5f)*(tx - 7.5f) / 128.0f);
	buffer[idx] = 0.0f;
	__syncthreads();

	// Compute angles and gradients
	float theta = 2.0f*3.1415f / 360.0f*d_sift[bx].orientation;
	float sina = sinf(theta);           // cosa -sina
	float cosa = cosf(theta);           // sina  cosa
	float scale = 12.0f / 16.0f*d_sift[bx].scale;
	float ssina = scale*sina;
	float scosa = scale*cosa;

	for (int y = ty; y<16; y += 8) {
		float xpos = d_sift[bx].xpos + (tx - 7.5f)*scosa - (y - 7.5f)*ssina;
		float ypos = d_sift[bx].ypos + (tx - 7.5f)*ssina + (y - 7.5f)*scosa;
		float dx = tex2D<float>(texObj, xpos + cosa, ypos + sina) -
			tex2D<float>(texObj, xpos - cosa, ypos - sina);
		float dy = tex2D<float>(texObj, xpos - sina, ypos + cosa) -
			tex2D<float>(texObj, xpos + sina, ypos - cosa);
		float grad = gauss[y] * gauss[tx] * sqrtf(dx*dx + dy*dy);
		float angf = 4.0f / 3.1415f*atan2f(dy, dx) + 4.0f;

		int hori = (tx + 2) / 4 - 1;      // Convert from (tx,y,angle) to bins      
		float horf = (tx - 1.5f) / 4.0f - hori;
		float ihorf = 1.0f - horf;
		int veri = (y + 2) / 4 - 1;
		float verf = (y - 1.5f) / 4.0f - veri;
		float iverf = 1.0f - verf;
		int angi = angf;
		int angp = (angi<7 ? angi + 1 : 0);
		angf -= angi;
		float iangf = 1.0f - angf;

		int hist = 8 * (4 * veri + hori);   // Each gradient measure is interpolated 
		int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
		int p2 = angp + hist;
		if (tx >= 2) {
			float grad1 = ihorf*grad;
			if (y >= 2) {   // Upper left
				float grad2 = iverf*grad1;
				atomicAdd(buffer + p1, iangf*grad2);
				atomicAdd(buffer + p2, angf*grad2);
			}
			if (y <= 13) {  // Lower left
				float grad2 = verf*grad1;
				atomicAdd(buffer + p1 + 32, iangf*grad2);
				atomicAdd(buffer + p2 + 32, angf*grad2);
			}
		}
		if (tx <= 14) {
			float grad1 = horf*grad;
			if (y >= 2) {    // Upper right
				float grad2 = iverf*grad1;
				atomicAdd(buffer + p1 + 8, iangf*grad2);
				atomicAdd(buffer + p2 + 8, angf*grad2);
			}
			if (y <= 13) {   // Lower right
				float grad2 = verf*grad1;
				atomicAdd(buffer + p1 + 40, iangf*grad2);
				atomicAdd(buffer + p2 + 40, angf*grad2);
			}
		}
	}
	__syncthreads();

	// Normalize twice and suppress peaks first time
	if (idx<64)
		sums[idx] = buffer[idx] * buffer[idx] + buffer[idx + 64] * buffer[idx + 64];
	__syncthreads();
	if (idx<32) sums[idx] = sums[idx] + sums[idx + 32];
	__syncthreads();
	if (idx<16) sums[idx] = sums[idx] + sums[idx + 16];
	__syncthreads();
	if (idx<8)  sums[idx] = sums[idx] + sums[idx + 8];
	__syncthreads();
	if (idx<4)  sums[idx] = sums[idx] + sums[idx + 4];
	__syncthreads();
	float tsum1 = sums[0] + sums[1] + sums[2] + sums[3];
	buffer[idx] = buffer[idx] * rsqrtf(tsum1);

	if (buffer[idx]>0.2f)
		buffer[idx] = 0.2f;
	__syncthreads();
	if (idx<64)
		sums[idx] = buffer[idx] * buffer[idx] + buffer[idx + 64] * buffer[idx + 64];
	__syncthreads();
	if (idx<32) sums[idx] = sums[idx] + sums[idx + 32];
	__syncthreads();
	if (idx<16) sums[idx] = sums[idx] + sums[idx + 16];
	__syncthreads();
	if (idx<8)  sums[idx] = sums[idx] + sums[idx + 8];
	__syncthreads();
	if (idx<4)  sums[idx] = sums[idx] + sums[idx + 4];
	__syncthreads();
	float tsum2 = sums[0] + sums[1] + sums[2] + sums[3];

	float *desc = d_sift[bx].data;
	desc[idx] = buffer[idx] * rsqrtf(tsum2);
	if (idx == 0) {
		d_sift[bx].xpos *= subsampling;
		d_sift[bx].ypos *= subsampling;
		d_sift[bx].scale *= subsampling;
	}
}


__global__ void ComputeOrientations(hipTextureObject_t texObj, SiftPoint *d_Sift, int fstPts)
{
	__shared__ float hist[64];
	__shared__ float gauss[11];
	const int tx = threadIdx.x;
	const int bx = blockIdx.x + fstPts;
	float i2sigma2 = -1.0f / (4.5f*d_Sift[bx].scale*d_Sift[bx].scale);
	if (tx<11)
		gauss[tx] = exp(i2sigma2*(tx - 5)*(tx - 5));
	if (tx<64)
		hist[tx] = 0.0f;
	__syncthreads();
	float xp = d_Sift[bx].xpos - 5.0f;
	float yp = d_Sift[bx].ypos - 5.0f;
	int yd = tx / 11;
	int xd = tx - yd * 11;
	float xf = xp + xd;
	float yf = yp + yd;
	if (yd<11) {
		float dx = tex2D<float>(texObj, xf + 1.0, yf) - tex2D<float>(texObj, xf - 1.0, yf);
		float dy = tex2D<float>(texObj, xf, yf + 1.0) - tex2D<float>(texObj, xf, yf - 1.0);
		int bin = 16.0f*atan2f(dy, dx) / 3.1416f + 16.5f;
		if (bin>31)
			bin = 0;
		float grad = sqrtf(dx*dx + dy*dy);
		atomicAdd(&hist[bin], grad*gauss[xd] * gauss[yd]);
	}
	__syncthreads();
	int x1m = (tx >= 1 ? tx - 1 : tx + 31);
	int x1p = (tx <= 30 ? tx + 1 : tx - 31);
	if (tx<32) {
		int x2m = (tx >= 2 ? tx - 2 : tx + 30);
		int x2p = (tx <= 29 ? tx + 2 : tx - 30);
		hist[tx + 32] = 6.0f*hist[tx] + 4.0f*(hist[x1m] + hist[x1p]) + (hist[x2m] + hist[x2p]);
	}
	__syncthreads();
	if (tx<32) {
		float v = hist[32 + tx];
		hist[tx] = (v>hist[32 + x1m] && v >= hist[32 + x1p] ? v : 0.0f);
	}
	__syncthreads();
	if (tx == 0) {
		float maxval1 = 0.0;
		float maxval2 = 0.0;
		int i1 = -1;
		int i2 = -1;
		for (int i = 0; i<32; i++) {
			float v = hist[i];
			if (v>maxval1) {
				maxval2 = maxval1;
				maxval1 = v;
				i2 = i1;
				i1 = i;
			}
			else if (v>maxval2) {
				maxval2 = v;
				i2 = i;
			}
		}
		float val1 = hist[32 + ((i1 + 1) & 31)];
		float val2 = hist[32 + ((i1 + 31) & 31)];
		float peak = i1 + 0.5f*(val1 - val2) / (2.0f*maxval1 - val1 - val2);
		d_Sift[bx].orientation = 11.25f*(peak<0.0f ? peak + 32.0f : peak);
		if (maxval2>0.8f*maxval1 && false) {
			float val1 = hist[32 + ((i2 + 1) & 31)];
			float val2 = hist[32 + ((i2 + 31) & 31)];
			float peak = i2 + 0.5f*(val1 - val2) / (2.0f*maxval2 - val1 - val2);
			unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
			if (idx<d_MaxNumPoints) {
				d_Sift[idx].xpos = d_Sift[bx].xpos;
				d_Sift[idx].ypos = d_Sift[bx].ypos;
				d_Sift[idx].scale = d_Sift[bx].scale;
				d_Sift[idx].sharpness = d_Sift[bx].sharpness;
				d_Sift[idx].edgeness = d_Sift[bx].edgeness;
				d_Sift[idx].orientation = 11.25f*(peak<0.0f ? peak + 32.0f : peak);;
				d_Sift[idx].subsampling = d_Sift[bx].subsampling;
			}
		}
	}
}

///////////////////////////////////////////////////////////////////////////////
// Subtract two images (multi-scale version)
///////////////////////////////////////////////////////////////////////////////

__global__ void FindPointsMulti(float *d_Data0, SiftPoint *d_Sift, int width, int pitch, int height, int nScales, float subsampling)
{
#define MEMWID (MINMAX_W + 2)
	__shared__ float ymin1[MEMWID], ymin2[MEMWID], ymin3[MEMWID];
	__shared__ float ymax1[MEMWID], ymax2[MEMWID], ymax3[MEMWID];
	__shared__ unsigned int cnt;
	__shared__ unsigned short points[96];

	int tx = threadIdx.x;
	int block = blockIdx.x / nScales;
	int scale = blockIdx.x - nScales*block;
	int minx = block*MINMAX_W;
	int maxx = min(minx + MINMAX_W, width);
	int xpos = minx + tx;
	int size = pitch*height;
	int ptr = size*scale + max(min(xpos - 1, width - 1), 0);

	if (tx == 0)
		cnt = 0;
	__syncthreads();

	int yloops = min(height - MINMAX_H*blockIdx.y, MINMAX_H);
	for (int y = 0; y<yloops; y++) {

		int ypos = MINMAX_H*blockIdx.y + y;
		int yptr0 = ptr + max(0, ypos - 1)*pitch;
		int yptr1 = ptr + ypos*pitch;
		int yptr2 = ptr + min(height - 1, ypos + 1)*pitch;
		{
			float d10 = d_Data0[yptr0];
			float d11 = d_Data0[yptr1];
			float d12 = d_Data0[yptr2];
			ymin1[tx] = fminf(fminf(d10, d11), d12);
			ymax1[tx] = fmaxf(fmaxf(d10, d11), d12);
		}
	{
		float d30 = d_Data0[yptr0 + 2 * size];
		float d31 = d_Data0[yptr1 + 2 * size];
		float d32 = d_Data0[yptr2 + 2 * size];
		ymin3[tx] = fminf(fminf(d30, d31), d32);
		ymax3[tx] = fmaxf(fmaxf(d30, d31), d32);
	}
		float d20 = d_Data0[yptr0 + 1 * size];
		float d21 = d_Data0[yptr1 + 1 * size];
		float d22 = d_Data0[yptr2 + 1 * size];
		ymin2[tx] = fminf(fminf(ymin1[tx], fminf(fminf(d20, d21), d22)), ymin3[tx]);
		ymax2[tx] = fmaxf(fmaxf(ymax1[tx], fmaxf(fmaxf(d20, d21), d22)), ymax3[tx]);
		__syncthreads();
		if (tx>0 && tx<MINMAX_W + 1 && xpos <= maxx) {
			if (d21<d_Threshold[1]) {
				float minv = fminf(fminf(fminf(ymin2[tx - 1], ymin2[tx + 1]), ymin1[tx]), ymin3[tx]);
				minv = fminf(fminf(minv, d20), d22);
				if (d21<minv) {
					int pos = atomicInc(&cnt, 31);
					points[3 * pos + 0] = xpos - 1;
					points[3 * pos + 1] = ypos;
					points[3 * pos + 2] = scale;
				}
			}
			if (d21>d_Threshold[0]) {
				float maxv = fmaxf(fmaxf(fmaxf(ymax2[tx - 1], ymax2[tx + 1]), ymax1[tx]), ymax3[tx]);
				maxv = fmaxf(fmaxf(maxv, d20), d22);
				if (d21>maxv) {
					int pos = atomicInc(&cnt, 31);
					points[3 * pos + 0] = xpos - 1;
					points[3 * pos + 1] = ypos;
					points[3 * pos + 2] = scale;
				}
			}
		}
		__syncthreads();
	}
	if (tx<cnt) {
		int xpos = points[3 * tx + 0];
		int ypos = points[3 * tx + 1];
		int scale = points[3 * tx + 2];
		int ptr = xpos + (ypos + (scale + 1)*height)*pitch;
		float val = d_Data0[ptr];
		float *data1 = &d_Data0[ptr];
		float dxx = 2.0f*val - data1[-1] - data1[1];
		float dyy = 2.0f*val - data1[-pitch] - data1[pitch];
		float dxy = 0.25f*(data1[+pitch + 1] + data1[-pitch - 1] - data1[-pitch + 1] - data1[+pitch - 1]);
		float tra = dxx + dyy;
		float det = dxx*dyy - dxy*dxy;
		if (tra*tra<d_EdgeLimit*det) {
			float edge = __fdividef(tra*tra, det);
			float dx = 0.5f*(data1[1] - data1[-1]);
			float dy = 0.5f*(data1[pitch] - data1[-pitch]);
			float *data0 = d_Data0 + ptr - height*pitch;
			float *data2 = d_Data0 + ptr + height*pitch;
			float ds = 0.5f*(data0[0] - data2[0]);
			float dss = 2.0f*val - data2[0] - data0[0];
			float dxs = 0.25f*(data2[1] + data0[-1] - data0[1] - data2[-1]);
			float dys = 0.25f*(data2[pitch] + data0[-pitch] - data2[-pitch] - data0[pitch]);
			float idxx = dyy*dss - dys*dys;
			float idxy = dys*dxs - dxy*dss;
			float idxs = dxy*dys - dyy*dxs;
			float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
			float idyy = dxx*dss - dxs*dxs;
			float idys = dxy*dxs - dxx*dys;
			float idss = dxx*dyy - dxy*dxy;
			float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
			float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
			float pds = idet*(idxs*dx + idys*dy + idss*ds);
			if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
				pdx = __fdividef(dx, dxx);
				pdy = __fdividef(dy, dyy);
				pds = __fdividef(ds, dss);
			}
			float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
			int maxPts = d_MaxNumPoints;
			unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
			idx = (idx >= maxPts ? maxPts - 1 : idx);
			d_Sift[idx].xpos = xpos + pdx;
			d_Sift[idx].ypos = ypos + pdy;
			d_Sift[idx].scale = d_Scales[scale] * exp2f(pds*d_Factor);
			d_Sift[idx].sharpness = val + dval;
			d_Sift[idx].edgeness = edge;
			d_Sift[idx].subsampling = subsampling;
		}
	}
}


__global__ void LaplaceMulti(hipTextureObject_t texObj, float *d_Result, int width, int pitch, int height)
{
	__shared__ float data1[(LAPLACE_W + 2 * LAPLACE_R)*LAPLACE_S];
	__shared__ float data2[LAPLACE_W*LAPLACE_S];
	const int tx = threadIdx.x;
	const int xp = blockIdx.x*LAPLACE_W + tx;
	const int yp = blockIdx.y;
	const int scale = threadIdx.y;
	float *kernel = d_Kernel2 + scale * 16;
	float *sdata1 = data1 + (LAPLACE_W + 2 * LAPLACE_R)*scale;
	float x = xp - 3.5;
	float y = yp + 0.5;
	sdata1[tx] = kernel[4] * tex2D<float>(texObj, x, y) +
		kernel[3] * (tex2D<float>(texObj, x, y - 1.0) + tex2D<float>(texObj, x, y + 1.0)) +
		kernel[2] * (tex2D<float>(texObj, x, y - 2.0) + tex2D<float>(texObj, x, y + 2.0)) +
		kernel[1] * (tex2D<float>(texObj, x, y - 3.0) + tex2D<float>(texObj, x, y + 3.0)) +
		kernel[0] * (tex2D<float>(texObj, x, y - 4.0) + tex2D<float>(texObj, x, y + 4.0));
	__syncthreads();
	float *sdata2 = data2 + LAPLACE_W*scale;
	if (tx<LAPLACE_W) {
		sdata2[tx] = kernel[4] * sdata1[tx + 4] +
			kernel[3] * (sdata1[tx + 3] + sdata1[tx + 5]) +
			kernel[2] * (sdata1[tx + 2] + sdata1[tx + 6]) +
			kernel[1] * (sdata1[tx + 1] + sdata1[tx + 7]) +
			kernel[0] * (sdata1[tx + 0] + sdata1[tx + 8]);
	}
	__syncthreads();
	if (tx<LAPLACE_W && scale<LAPLACE_S - 1 && xp<width)
		d_Result[scale*height*pitch + yp*pitch + xp] = sdata2[tx] - sdata2[tx + LAPLACE_W];
}


void InitCuda(int devNum)
{
	int nDevices;
	hipGetDeviceCount(&nDevices);
	if (!nDevices) {
		std::cerr << "No CUDA devices available" << std::endl;
		return;
	}
	devNum = std::min(nDevices - 1, devNum);
	deviceInit(devNum);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devNum);
	printf("Device Number: %d\n", devNum);
	printf("  Device name: %s\n", prop.name);
	printf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate / 1000);
	printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %.1f\n\n",
		2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
}

void ExtractSift(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, float subsampling)
{
	TimerGPU timer(0);
	int totPts = 0;
	safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_PointCounter), &totPts, sizeof(int)));
	safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_MaxNumPoints), &siftData.maxPts, sizeof(int)));

	const int nd = NUM_SCALES + 3;
	int w = img.width;
	int h = img.height;
	int p = iAlignUp(w, 128);
	int size = 0;         // image sizes
	int sizeTmp = nd*h*p; // laplace buffer sizes
	for (int i = 0; i<numOctaves; i++) {
		w /= 2;
		h /= 2;
		int p = iAlignUp(w, 128);
		size += h*p;
		sizeTmp += nd*h*p;
	}
	float *memoryTmp = NULL;
	size_t pitch;
	size += sizeTmp;
	safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size + 4095) / 4096 * sizeof(float)));
	float *memorySub = memoryTmp + sizeTmp;

	ExtractSiftLoop(siftData, img, numOctaves, initBlur, thresh, lowestScale, subsampling, memoryTmp, memorySub);
	safeCall(hipMemcpyFromSymbol(&siftData.numPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
	siftData.numPts = (siftData.numPts<siftData.maxPts ? siftData.numPts : siftData.maxPts);
	safeCall(hipFree(memoryTmp));
#ifdef MANAGEDMEM
	safeCall(hipDeviceSynchronize());
#else
	if (siftData.h_data)
		safeCall(hipMemcpy(siftData.h_data, siftData.d_data, sizeof(SiftPoint)*siftData.numPts, hipMemcpyDeviceToHost));
#endif
	double totTime = timer.read();
#ifndef VERBOSE
	printf("Total time incl memory =      %.2f ms\n", totTime);
#endif
}

extern double DynamicMain(CudaImage &img, SiftData &siftData, int numOctaves, double initBlur, float thresh, float lowestScale, float edgeLimit, float *memoryTmp);

void ExtractSiftLoop(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp, float *memorySub)
{
	TimerGPU timer(0);
#if 1
	int w = img.width;
	int h = img.height;
	if (numOctaves>1) {
		CudaImage subImg;
		int p = iAlignUp(w / 2, 128);
		subImg.Allocate(w / 2, h / 2, p, false, memorySub);
		ScaleDown(subImg, img, 0.5f);
		float totInitBlur = (float)sqrt(initBlur*initBlur + 0.5f*0.5f) / 2.0f;
		ExtractSiftLoop(siftData, subImg, numOctaves - 1, totInitBlur, thresh, lowestScale, subsampling*2.0f, memoryTmp, memorySub + (h / 2)*p);
	}
	if (lowestScale<subsampling*2.0f)
		ExtractSiftOctave(siftData, img, initBlur, thresh, lowestScale, subsampling, memoryTmp);
#else
	DynamicMain(img, siftData, numOctaves, initBlur, thresh, lowestScale, 10.0f, memoryTmp);
#endif
	double totTime = timer.read();
#ifdef VERBOSE
	printf("ExtractSift time total =      %.2f ms\n\n", totTime);
#endif
}

void ExtractSiftOctave(SiftData &siftData, CudaImage &img, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp)
{
	const int nd = NUM_SCALES + 3;
	TimerGPU timer0;
	CudaImage diffImg[nd];
	int w = img.width;
	int h = img.height;
	int p = iAlignUp(w, 128);
	for (int i = 0; i<nd - 1; i++)
		diffImg[i].Allocate(w, h, p, false, memoryTmp + i*p*h);

	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = img.d_data;
	resDesc.res.pitch2D.width = img.width;
	resDesc.res.pitch2D.height = img.height;
	resDesc.res.pitch2D.pitchInBytes = img.pitch*sizeof(float);
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
	// Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;
	// Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	TimerGPU timer1;
	float baseBlur = pow(2.0f, -1.0f / NUM_SCALES);
	float diffScale = pow(2.0f, 1.0f / NUM_SCALES);
	LaplaceMulti(texObj, diffImg, baseBlur, diffScale, initBlur);
	int fstPts = 0;
	safeCall(hipMemcpyFromSymbol(&fstPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
	double sigma = baseBlur*diffScale;
	FindPointsMulti(diffImg, siftData, thresh, 10.0f, sigma, 1.0f / NUM_SCALES, lowestScale / subsampling, subsampling);
	double gpuTimeDoG = timer1.read();
	TimerGPU timer4;
	int totPts = 0;
	safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
	totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
	if (totPts>fstPts) {
		ComputeOrientations(texObj, siftData, fstPts, totPts);
		safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
		totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
		ExtractSiftDescriptors(texObj, siftData, fstPts, totPts, subsampling);
	}
	safeCall(hipDestroyTextureObject(texObj));
	double gpuTimeSift = timer4.read();

	double totTime = timer0.read();
#ifdef VERBOSE
	printf("GPU time : %.2f ms + %.2f ms + %.2f ms = %.2f ms\n", totTime - gpuTimeDoG - gpuTimeSift, gpuTimeDoG, gpuTimeSift, totTime);
	safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
	totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
	if (totPts>0)
		printf("           %.2f ms / DoG,  %.4f ms / Sift,  #Sift = %d\n", gpuTimeDoG / NUM_SCALES, gpuTimeSift / (totPts - fstPts), totPts - fstPts);
#endif
}

void InitSiftData(SiftData &data, int num, bool host, bool dev)
{
	data.numPts = 0;
	data.maxPts = num;
	int sz = sizeof(SiftPoint)*num;
#ifdef MANAGEDMEM
	safeCall(hipMallocManaged((void **)&data.m_data, sz));
#else
	data.h_data = NULL;
	if (host)
		data.h_data = (SiftPoint *)malloc(sz);
	data.d_data = NULL;
	if (dev)
		safeCall(hipMalloc((void **)&data.d_data, sz));
#endif
}

void FreeSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
	safeCall(hipFree(data.m_data));
#else
	if (data.d_data != NULL)
		safeCall(hipFree(data.d_data));
	data.d_data = NULL;
	if (data.h_data != NULL)
		free(data.h_data);
#endif
	data.numPts = 0;
	data.maxPts = 0;
}

void PrintSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
	SiftPoint *h_data = data.m_data;
#else
	SiftPoint *h_data = data.h_data;
	if (data.h_data == NULL) {
		h_data = (SiftPoint *)malloc(sizeof(SiftPoint)*data.maxPts);
		safeCall(hipHostMalloc((void **)&h_data, sizeof(SiftPoint)*data.maxPts));
		safeCall(hipMemcpy(h_data, data.d_data, sizeof(SiftPoint)*data.numPts, hipMemcpyDeviceToHost));
		data.h_data = h_data;
	}
#endif
	for (int i = 0; i<data.numPts; i++) {
		printf("xpos         = %.2f\n", h_data[i].xpos);
		printf("ypos         = %.2f\n", h_data[i].ypos);
		printf("scale        = %.2f\n", h_data[i].scale);
		printf("sharpness    = %.2f\n", h_data[i].sharpness);
		printf("edgeness     = %.2f\n", h_data[i].edgeness);
		printf("orientation  = %.2f\n", h_data[i].orientation);
		printf("score        = %.2f\n", h_data[i].score);
		float *siftData = (float*)&h_data[i].data;
		for (int j = 0; j<8; j++) {
			if (j == 0)
				printf("data = ");
			else
				printf("       ");
			for (int k = 0; k<16; k++)
				if (siftData[j + 8 * k]<0.05)
					printf(" .   ");
				else
					printf("%.2f ", siftData[j + 8 * k]);
			printf("\n");
		}
	}
	printf("Number of available points: %d\n", data.numPts);
	printf("Number of allocated points: %d\n", data.maxPts);
}

///////////////////////////////////////////////////////////////////////////////
// Host side master functions
///////////////////////////////////////////////////////////////////////////////

double ScaleDown(CudaImage &res, CudaImage &src, float variance)
{
	if (res.d_data == NULL || src.d_data == NULL) {
		printf("ScaleDown: missing data\n");
		return 0.0;
	}
	float h_Kernel[5];
	float kernelSum = 0.0f;
	for (int j = 0; j<5; j++) {
		h_Kernel[j] = (float)expf(-(double)(j - 2)*(j - 2) / 2.0 / variance);
		kernelSum += h_Kernel[j];
	}
	for (int j = 0; j<5; j++)
		h_Kernel[j] /= kernelSum;
	safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel1), h_Kernel, 5 * sizeof(float)));
	dim3 blocks(iDivUp(src.width, SCALEDOWN_W), iDivUp(src.height, SCALEDOWN_H));
	dim3 threads(SCALEDOWN_W + 4);
	ScaleDown << <blocks, threads >> >(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch);
	checkMsg("ScaleDown() execution failed\n");
	return 0.0;
}

double ComputeOrientations(hipTextureObject_t texObj, SiftData &siftData, int fstPts, int totPts)
{
	dim3 blocks(totPts - fstPts);
	dim3 threads(128);
#ifdef MANAGEDMEM
	ComputeOrientations << <blocks, threads >> >(texObj, siftData.m_data, fstPts);
#else
	ComputeOrientations << <blocks, threads >> >(texObj, siftData.d_data, fstPts);
#endif
	checkMsg("ComputeOrientations() execution failed\n");
	return 0.0;
}

double ExtractSiftDescriptors(hipTextureObject_t texObj, SiftData &siftData, int fstPts, int totPts, float subsampling)
{
	dim3 blocks(totPts - fstPts);
	dim3 threads(16, 8);
#ifdef MANAGEDMEM
	ExtractSiftDescriptors << <blocks, threads >> >(texObj, siftData.m_data, fstPts, subsampling);
#else
	ExtractSiftDescriptors << <blocks, threads >> >(texObj, siftData.d_data, fstPts, subsampling);

#endif
	checkMsg("ExtractSiftDescriptors() execution failed\n");
	return 0.0;
}

//==================== Multi-scale functions ===================//

double LaplaceMulti(hipTextureObject_t texObj, CudaImage *results, float baseBlur, float diffScale, float initBlur)
{
	float kernel[12 * 16];
	float scale = baseBlur;
	for (int i = 0; i<NUM_SCALES + 3; i++) {
		float kernelSum = 0.0f;
		float var = scale*scale - initBlur*initBlur;
		for (int j = -LAPLACE_R; j <= LAPLACE_R; j++) {
			kernel[16 * i + j + LAPLACE_R] = (float)expf(-(double)j*j / 2.0 / var);
			kernelSum += kernel[16 * i + j + LAPLACE_R];
		}
		for (int j = -LAPLACE_R; j <= LAPLACE_R; j++)
			kernel[16 * i + j + LAPLACE_R] /= kernelSum;
		scale *= diffScale;
	}
	safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel2), kernel, 12 * 16 * sizeof(float)));
	int width = results[0].width;
	int pitch = results[0].pitch;
	int height = results[0].height;
	dim3 blocks(iDivUp(width + 2 * LAPLACE_R, LAPLACE_W), height);
	dim3 threads(LAPLACE_W + 2 * LAPLACE_R, LAPLACE_S);
	LaplaceMulti << <blocks, threads >> >(texObj, results[0].d_data, width, pitch, height);
	checkMsg("LaplaceMulti() execution failed\n");
	return 0.0;
}

double FindPointsMulti(CudaImage *sources, SiftData &siftData, float thresh, float edgeLimit, float scale, float factor, float lowestScale, float subsampling)
{
	if (sources->d_data == NULL) {
		printf("FindPointsMulti: missing data\n");
		return 0.0;
	}
	int w = sources->width;
	int p = sources->pitch;
	int h = sources->height;
	float threshs[2] = { thresh, -thresh };
	float scales[NUM_SCALES];
	float diffScale = pow(2.0f, factor);
	for (int i = 0; i<NUM_SCALES; i++) {
		scales[i] = scale;
		scale *= diffScale;
	}
	safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Threshold), &threshs, 2 * sizeof(float)));
	safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_EdgeLimit), &edgeLimit, sizeof(float)));
	safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Scales), scales, sizeof(float)*NUM_SCALES));
	safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Factor), &factor, sizeof(float)));

	dim3 blocks(iDivUp(w, MINMAX_W)*NUM_SCALES, iDivUp(h, MINMAX_H));
	dim3 threads(MINMAX_W + 2);
#ifdef MANAGEDMEM
	FindPointsMulti << <blocks, threads >> >(sources->d_data, siftData.m_data, w, p, h, NUM_SCALES, subsampling);
#else
	FindPointsMulti << <blocks, threads >> >(sources->d_data, siftData.d_data, w, p, h, NUM_SCALES, subsampling);
#endif
	checkMsg("FindPointsMulti() execution failed\n");
	return 0.0;
}

//**********************some IO ********************************
void tk_write_Sift_Mat(SiftData* m_sift, const std::string& dumpname){
	
	//
	if (m_sift == NULL && m_sift->numPts && m_sift->h_data) {
		printf("sift feature if empty!\n"); return;
	}

	//




}